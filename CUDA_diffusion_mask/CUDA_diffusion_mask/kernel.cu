#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <chrono>
#include <vector>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb_image_resize.h"

#define FREE_MEMORY \
    do { \
        return 0; \
    } while(0);

#define CUDA_CALL_AND_CHECK(call, msg) \
    do { \
        cuda_error = call; \
        if (cuda_error != hipSuccess) { \
            printf("Example FAILED: CUDA API returned error = %d, details: " #msg "\n", cuda_error); \
            FREE_MEMORY; \
            return -1; \
        } \
    } while(0);

void print_arr(double arr[], int size);
void print_arr(int arr[], int size);
void print_arr_3d(double arr[], int nx, int ny, int nz);
int write_to_csv(double arr[], int nx, int ny, int nz, const char* filename);
void print_debug(double* a, double* b, double* c, double* x, double* y, int size);

void setAx(double Ax_a[], double Ax_b[], double Ax_c[], int nx, int ny, int nz, double sigma_x);
void setAy(double Ay_a[], double Ay_b[], double Ay_c[], int mask[], int nx, int ny, int nz, double sigma_y);
void setAz(double Az_a[], double Az_b[], double Az_c[], int nx, int ny, int nz, double sigma_z);

void fixEdges(double C[], int nx, int ny, int nz);

__global__ void generate_bx(double* C, double* bx, int nx, int ny, int nz);
__global__ void generate_by(double* C, double* Cx, double* by, int* mask, int nx, int ny, int nz);
__global__ void generate_bz(double* C, double* Cy, double* bz, int nx, int ny, int nz);
__global__ void update_C(double* C, double* Cz, int* mask, int nx, int ny, int nz);

void solve_cr(double* d_a, double* d_b, double* d_c, double* d_x, double* d_y, int size);
__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size);
__global__ void elimination(double* a, double* b, double* c, double* x, double* y, int depth, int size_padded, int size);
__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size);

int loadImage(const char* filename, int real_width, int real_height, int* mask);
int readCSVFlattened(const std::string & filename, int* mask, int width, int height);
int check_double(char* in, double* out);
int check_and_apply(double* output, char* D, char* T, char* dt, char* l, char* n, char* c0);
int check_and_apply(double* output, char* D, char* T, char* dt, char* lx, char* ly, char* lz, char* nx, char* ny, char* nz, char* c0);
void print_progress_bar(int i, int nt);

const int block_size = 512;
__device__ double device_constant[4];

hipError_t cuda_error = hipSuccess;

int main(int argc, char* argv[])
{
    double D, T, dt, lx, ly, lz, c0;
    int nx, ny, nz;
    double* output = NULL;

    // process constants
    if (argc == 9) {
        output = new double[6];
        check_and_apply(output, argv[3], argv[4], argv[5], argv[6], argv[7], argv[8]);
        D = output[0];
        T = output[1];
        dt = output[2];
        lx = output[3];
        ly = output[3];
        lz = output[3];
        nx = (int)output[4];
        ny = (int)output[4];
        nz = (int)output[4];
        c0 = output[5];
        printf("Parameter:\nD:%f\nT:%f\ndt:%f\nl:%f\nn:%d\nc0:%f\n", D, T, dt, lx, nx, c0);
    } 
    else if (argc == 13) {
        output = new double[10];
        check_and_apply(output, argv[3], argv[4], argv[5], argv[6], argv[7], argv[8], argv[9], argv[10], argv[11], argv[12]);
        D = output[0];
        T = output[1];
        dt = output[2];
        lx = output[3];
        ly = output[4];
        lz = output[5];
        nx = (int)output[6];
        ny = (int)output[7];
        nz = (int)output[8];
        c0 = output[9];
        printf("Parameter:\nD:%f\nT:%f\ndt:%f\nlx:%f\nly:%f\nlz:%f\nnx:%d\nny:%d\nnz:%d\nc0:%f\n", D, T, dt, lx, ly, lz, nx, ny, nz, c0);
    }
    else {
        fprintf(stderr, "ERROR, ARGUMENT COUNT %d INCORRECT\n", argc);
        return 1;
    }

    // process file input
    const char* filename;
    filename = argv[1];

    const char* dot = strrchr(filename, '.');
    printf("File type: %s\n", dot);

    // initialize mask array
    int real_width = (nx - 2);
    int real_height = (nz - 2);
    int* mask = new int[real_height * real_width];

    if (strcmp(dot + 1, "csv") == 0) {
        int err = readCSVFlattened(filename, mask, real_width, real_height);
        if (err == 1) {
            fprintf(stderr, "ERROR, CANT OPEN FILE\n");
            return err;
        }
    }
    else if (strcmp(dot + 1, "png") == 0) {
        int err = loadImage(filename, real_width, real_height, mask);
        if (err == 1) {
            fprintf(stderr, "ERROR, CANT OPEN FILE\n");
            return err;
        }
    }
    else {
        fprintf(stderr, "ERROR, FILE NOT SUPPORTED\n");
        return 1;
    }

    const char* output_name = strcat(argv[2], ".csv");
    printf("Output: %s\n", output_name);

    // calculate parameter needed for simulation
    const int nt = int(T / dt);
    const double dx = lx / (nx - 1);
    const double dy = ly / (ny - 1);
    const double dz = lz / (nz - 1);
    const double sigma_x = D * dt / pow(dx, 2);
    const double sigma_y = D * dt / pow(dy, 2);
    const double sigma_z = D * dt / pow(dz, 2);

    const int size = (nx - 2) * (ny - 2) * (nz - 2); // matrix size harus 2^n

    // move constants to device
    double h_array[4] = { sigma_x, sigma_y, sigma_z, c0 };
    hipMemcpyToSymbol(HIP_SYMBOL(device_constant), h_array, sizeof(h_array));

    // initial condition
    double* C = (double*)calloc(nx * ny * nz, sizeof(double));

    for (int z = 1; z < nz - 1; z++)
    {
        for (int x = 1; x < nx - 1; x++)
        {
            int y = ny - 1;
            int idx = z * ny * nx + y * nx + x;
            int id_mask = (z - 1) * (nx - 2) + (x - 1);
            C[idx] = c0 * (1 - mask[id_mask]);
        }
    }

    // allocate memory on device
    double* Ax_a = (double*)calloc(size, sizeof(double));
    double* Ax_b = (double*)calloc(size, sizeof(double));
    double* Ax_c = (double*)calloc(size, sizeof(double));

    double* Ay_a = (double*)calloc(size, sizeof(double));
    double* Ay_b = (double*)calloc(size, sizeof(double));
    double* Ay_c = (double*)calloc(size, sizeof(double));

    double* Az_a = (double*)calloc(size, sizeof(double));
    double* Az_b = (double*)calloc(size, sizeof(double));
    double* Az_c = (double*)calloc(size, sizeof(double));

    // initialize Ax, Ay, Az
    setAx(Ax_a, Ax_b, Ax_c, nx, ny, nz, sigma_x);
    setAy(Ay_a, Ay_b, Ay_c, mask, nx, ny, nz, sigma_y);
    setAz(Az_a, Az_b, Az_c, nx, ny, nz, sigma_z);

    double* d_Ax_a, * d_Ax_b, * d_Ax_c;
    double* d_Ay_a, * d_Ay_b, * d_Ay_c;
    double* d_Az_a, * d_Az_b, * d_Az_c;
    double* d_bx, * d_by, * d_bz;
    double* d_Cx, * d_Cy, * d_Cz, * d_C;
    int* d_mask;

    //allocate memory on device
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ax_a, (size) * sizeof(double)), "d_Ax_a");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ax_b, (size) * sizeof(double)), "d_Ax_b");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ax_c, (size) * sizeof(double)), "d_Ax_c");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ay_a, (size) * sizeof(double)), "d_Ay_a");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ay_b, (size) * sizeof(double)), "d_Ay_b");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ay_c, (size) * sizeof(double)), "d_Ay_c");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Az_a, (size) * sizeof(double)), "d_Az_a");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Az_b, (size) * sizeof(double)), "d_Az_b");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Az_c, (size) * sizeof(double)), "d_Az_c");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_bx, (size) * sizeof(double)), "d_bx");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_by, (size) * sizeof(double)), "d_by");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_bz, (size) * sizeof(double)), "d_bz");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Cx, (size) * sizeof(double)), "d_Cx");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Cy, (size) * sizeof(double)), "d_Cy");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Cz, (size) * sizeof(double)), "d_Cz");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_C, (nx * ny * nz) * sizeof(double)), "d_C");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_mask, (nz - 2) * (nx - 2) * sizeof(int)), "d_mask");

    // copy host memory to device
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ax_a, Ax_a, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ax_b, Ax_b, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ax_c, Ax_c, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ay_a, Ay_a, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ay_b, Ay_b, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ay_c, Ay_c, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Az_a, Az_a, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Az_b, Az_b, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Az_c, Az_c, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_C, C, (nx * ny * nz) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_mask, mask, (nz - 2) * (nx - 2) * sizeof(int), hipMemcpyHostToDevice), "");

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    printf("SIMULATING...\n");
    // start simulation
    for (int i = 0; i < nt; i++)
    {
        int block_count = (int)ceil(float(size) / block_size);

        generate_bx<<<block_count, block_size>>>(d_C, d_bx, nx, ny, nz);
        //print_debug(d_Ax_a, d_Ax_b, d_Ax_c, d_Cx, d_bx, size);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for C'
        solve_cr(d_Ax_a, d_Ax_b, d_Ax_c, d_Cx, d_bx, size);

        generate_by<<<block_count, block_size>>>(d_C, d_Cx, d_by, d_mask, nx, ny, nz);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for C''
        solve_cr(d_Ay_a, d_Ay_b, d_Ay_c, d_Cy, d_by, size);

        generate_bz<<<block_count, block_size>>>(d_C, d_Cy, d_bz, nx, ny, nz);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for Ct+1
        solve_cr(d_Az_a, d_Az_b, d_Az_c, d_Cz, d_bz, size);

        update_C<<<block_count, block_size>>>(d_C, d_Cz, d_mask, nx, ny, nz);
        hipDeviceSynchronize();

        print_progress_bar(i+1, nt);
    }
    printf("\n");
    CUDA_CALL_AND_CHECK(hipMemcpy(C, d_C, (nx * ny * nz) * sizeof(double), hipMemcpyDeviceToHost), "");

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
    
    fixEdges(C, nx, ny, nz);
    //print_arr_3d(C, n);
    int err = write_to_csv(C, nx, ny, nz, output_name);
    if (err == 1) { return err; }
    return 0;
}

void print_arr(double arr[], int size)
{
    printf("[");
    for (int i = 0; i < size - 1; i++)
    {
        printf("%f, ", arr[i]);
    }
    printf("%f", arr[size - 1]);
    printf("]\n");
}

void print_arr(int arr[], int size)
{
    printf("[");
    for (int i = 0; i < size - 1; i++)
    {
        printf("%d, ", arr[i]);
    }
    printf("%d", arr[size - 1]);
    printf("]\n");
}

int write_to_csv(double arr[], int nx, int ny, int nz, const char* filename) {
    std::ofstream myfile;
    myfile.open(filename);

    if (!myfile) {
        fprintf(stderr, "ERROR, CANT CREATE/OPEN FILE\n");
        return 1;
    }

    for (int z = 0; z < nz; z++)
    {
        for (int y = 0; y < ny; y++)
        {
            for (int x = 0; x < nx; x++)
            {
                myfile << arr[z * ny * nx + y * nx + x] << ",";
            }
        }
    }
    myfile << nx << "," << ny << "," << nz;
    myfile.close();
}

void print_debug(double* d_a, double* d_b, double* d_c, double* d_x, double* d_y, int size)
{
    double* a = new double[size];
    double* b = new double[size];
    double* c = new double[size];
    double* x = new double[size];
    double* y = new double[size];

    hipMemcpy(a, d_a, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(x, d_x, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, (size) * sizeof(double), hipMemcpyDeviceToHost);

    printf("----------------------\n");
    print_arr(a, size);
    print_arr(b, size);
    print_arr(c, size);
    print_arr(x, size);
    print_arr(y, size);
    printf("----------------------\n");

    delete[]a;
    delete[]b;
    delete[]c;
    delete[]x;
    delete[]y;
}

void setAx(double Ax_a[], double Ax_b[], double Ax_c[], int nx, int ny, int nz, double sigma_x)
{
    for (int z = 0; z < nz - 2; z++)
    {
        for (int y = 0; y < ny - 2; y++)
        {
            for (int x = 0; x < nx - 2; x++)
            {
                int idx = z * (ny - 2) * (nx - 2) + y * (nx - 2) + x;

                if (x == 0) {
                    Ax_c[idx] = -0.5 * sigma_x;
                    Ax_b[idx] = 1 + (0.5 * sigma_x);
                }
                else if (x == nx - 3) {
                    Ax_a[idx] = -0.5 * sigma_x;
                    Ax_b[idx] = 1 + (0.5 * sigma_x);
                }
                else {
                    Ax_c[idx] = -0.5 * sigma_x;
                    Ax_b[idx] = (1.0 + sigma_x);
                    Ax_a[idx] = -0.5 * sigma_x;
                }
            }
        }
    }
}

void setAy(double Ay_a[], double Ay_b[], double Ay_c[], int mask[], int nx, int ny, int nz, double sigma_y)
{
    for (int z = 0; z < nz - 2; z++)
    {
        for (int y = 0; y < ny - 2; y++)
        {
            for (int x = 0; x < nx - 2; x++)
            {
                int idy = z * (ny - 2) * (nx - 2) + x * (ny - 2) + y;
                int id_mask = z * (nx - 2) + x;

                if (y == 0) {
                    Ay_c[idy] = -0.5 * sigma_y;
                    Ay_b[idy] = 1.0 + (0.5 * sigma_y);
                }
                else if (y == ny - 3) {
                    Ay_a[idy] = -0.5 * sigma_y;
                    if (mask[id_mask] == 1) {
                        Ay_b[idy] = 1 + (0.5 * sigma_y);
                    }
                    else {
                        Ay_b[idy] = (1.0 + sigma_y);
                    }
                }
                else {
                    Ay_c[idy] = -0.5 * sigma_y;
                    Ay_b[idy] = (1.0 + sigma_y);
                    Ay_a[idy] = -0.5 * sigma_y;
                }
            }
        }
    }
}

void setAz(double Az_a[], double Az_b[], double Az_c[], int nx, int ny, int nz, double sigma_z)
{
    for (int z = 0; z < nz - 2; z++)
    {
        for (int y = 0; y < ny - 2; y++)
        {
            for (int x = 0; x < nx - 2; x++)
            {
                int idz = x * (ny - 2) * (nz - 2) + y * (nz - 2) + z;

                if (z == 0) {
                    Az_b[idz] = 1 + (0.5 * sigma_z);
                    Az_c[idz] = -0.5 * sigma_z;
                }
                else if (z == nz - 3) {
                    Az_a[idz] = -0.5 * sigma_z;
                    Az_b[idz] = 1 + (0.5 * sigma_z);
                }
                else {
                    Az_a[idz] = -0.5 * sigma_z;
                    Az_b[idz] = (1.0 + sigma_z);
                    Az_c[idz] = -0.5 * sigma_z;
                }
            }
        }
    }
}

void fixEdges(double C[], int nx, int ny, int nz)
{
    for (int x = 0; x < nx; x++) {
        C[(ny - 1) * nx + x] = C[ny * nx + (ny - 1) * nx + x];
        C[(nz - 1) * ny * nx + (ny - 1) * nx + x] = C[(nz - 2) * ny * nx + (ny - 1) * nx + x];
        C[x] = C[ny * nx + x];
        C[(nz - 1) * ny * nx + x] = C[(nz - 2) * ny * nx + x];
    }
    for (int y = 0; y < ny; y++) {

        C[y * nx] = C[y * nx + 1];
        C[y * nx + (nx - 1)] = C[y * nx + (nx - 2)];
        C[(nz - 1) * ny * nx + y * nx] = C[(nz - 1) * ny * nx + y * nx + 1];
        C[(nz - 1) * ny * nx + y * nx + (nx - 1)] = C[(nz - 1) * ny * nx + y * nx + (nx - 2)];
    }
    for (int z = 0; z < nz; z++) {

        C[z * ny * nx + (ny - 1) * nx] = C[z * ny * nx + (ny - 1) * nx + 1];
        C[z * ny * nx + (ny - 1) * nx + (nx - 1)] = C[z * ny * nx + (ny - 1) * nx + (nx - 2)];
        C[z * ny * nx] = C[z * ny * nx + 1];
        C[z * ny * nx + (nx - 1)] = C[z * ny * nx + (nx - 2)];
    }
}

__global__ void generate_bx(double* C, double* bx, int nx, int ny, int nz)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (nx - 2) * (ny - 2) * (nz - 2))
    {
        double sigma_x = device_constant[0];
        double sigma_y = device_constant[1];
        double sigma_z = device_constant[2];

        int x = i % (nx - 2);
        int y = (i / (nx - 2)) % (ny - 2);
        int z = i / ((nx - 2) * (ny - 2));

        int idx = i;
        int c_id = (z + 1) * ny * nx + (y + 1) * nx + (x + 1);

        int left = c_id - 1;
        int right = c_id + 1;
        int up = c_id + nx;
        int down = c_id - nx;
        int front = c_id + ny * nx;
        int back = c_id - ny * nx;

        bx[idx] = C[c_id] + 0.5 * sigma_x * (C[left] - 2 * C[c_id] + C[right])
            + sigma_y * (C[down] - 2 * C[c_id] + C[up])
            + sigma_z * (C[back] - 2 * C[c_id] + C[front]);
    }
}

__global__ void generate_by(double* C, double* Cx, double* by, int* mask, int nx, int ny, int nz)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (nx - 2) * (ny - 2) * (nz - 2))
    {
        double sigma_y = device_constant[1];
        double c0 = device_constant[3];

        int x = i % (nx - 2);
        int y = (i / (nx - 2)) % (ny - 2);
        int z = i / ((nx - 2) * (ny - 2));

        int idx = i;
        int idy = z * (ny - 2) * (nx - 2) + x * (ny - 2) + y;
        int id_mask = z * (nx - 2) + x;
        int c_id = (z + 1) * ny * nx + (y + 1) * nx + (x + 1);

        int up = c_id + nx;
        int down = c_id - nx;

        by[idy] = Cx[idx] - 0.5 * sigma_y * (C[down] - 2 * C[c_id] + C[up]);

        if (y == ny - 3)
        {
            if (mask[id_mask] == 0)
            {
                by[idy] += c0 * (0.5 * sigma_y);
            }
        }
    }
}

__global__ void generate_bz(double* C, double* Cy, double* bz, int nx, int ny, int nz)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (nx - 2) * (ny - 2) * (nz - 2))
    {
        double sigma_z = device_constant[2];

        int x = i % (nx - 2);
        int y = (i / (nx - 2)) % (ny - 2);
        int z = i / ((nx - 2) * (ny - 2));

        int idy = z * (ny - 2) * (nx - 2) + x * (ny - 2) + y;
        int idz = x * (ny - 2) * (nz - 2) + y * (nz - 2) + z;
        int c_id = (z + 1) * ny * nx + (y + 1) * nx + (x + 1);

        int front = c_id + ny * nx;
        int back = c_id - ny * nx;

        bz[idz] = Cy[idy] - 0.5 * sigma_z * (C[back] - 2 * C[c_id] + C[front]);
    }
}

__global__ void update_C(double* C, double* Cz, int* mask, int nx, int ny, int nz)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (nx - 2) * (ny - 2) * (nz - 2))
    {
        int x = i % (nx - 2);
        int y = (i / (nx - 2)) % (ny - 2);
        int z = i / ((nx - 2) * (ny - 2));

        int idz = x * (ny - 2) * (nz - 2) + y * (nz - 2) + z;
        int id_mask = z * (nx - 2) + x;
        int c_id = (z + 1) * ny * nx + (y + 1) * nx + (x + 1);

        C[c_id] = Cz[idz];

        int left = c_id - 1;
        int right = c_id + 1;
        int up = c_id + nx;
        int down = c_id - nx;
        int front = c_id + ny * nx;
        int back = c_id - ny * nx;

        if (x == 0) {
            C[left] = C[c_id];
        }
        if (x == nx - 3) {
            C[right] = C[c_id];
        }
        if (y == 0) {
            C[down] = C[c_id];
        }
        if (y == ny - 3) {
            if (mask[id_mask] == 1) {
                C[up] = C[c_id];
            }
        }
        if (z == 0) {
            C[back] = C[c_id];
        }
        if (z == nz - 3) {
            C[front] = C[c_id];
        }
    }
}

// input is array at global memory, output d_x
void solve_cr(double* d_a, double* d_b, double* d_c, double* d_x, double* d_y, int size)
{
    int size_padded = 1 << (int)(ceil(log2(size)));
    int depth = int(log2(size_padded)) - 1;

    // handle malloc at global memory
    double* dd_a, * dd_b, * dd_c, * dd_y;
    hipMalloc(&dd_a, (size) * sizeof(double));
    hipMalloc(&dd_b, (size) * sizeof(double));
    hipMalloc(&dd_c, (size) * sizeof(double));
    hipMalloc(&dd_y, (size) * sizeof(double));

    hipMemcpy(dd_a, d_a, (size) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_b, d_b, (size) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_c, d_c, (size) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_y, d_y, (size) * sizeof(double), hipMemcpyDeviceToDevice);

    // forward reduction
    for (int i = 0; i < depth; i++)
    {
        int thread_count = size_padded / (1 << (i + 1));
        int block_count = (thread_count + block_size - 1) / block_size;

        forward_reduce << <block_count, block_size >> > (dd_a, dd_b, dd_c, d_x, dd_y, i, thread_count, size);
        hipDeviceSynchronize();
    }

    // elimination
    elimination << <1, 1 >> > (dd_a, dd_b, dd_c, d_x, dd_y, depth, size_padded, size);
    hipDeviceSynchronize();

    // backward subtitution
    for (int i = depth - 1; i >= 0; i--)
    {
        int thread_count = size_padded / (1 << (i + 1));
        int block_count = (thread_count + block_size - 1) / block_size;

        backward_subs << <block_count, block_size >> > (dd_a, dd_b, dd_c, d_x, dd_y, i, thread_count, size);
        hipDeviceSynchronize();
    }

    hipFree(dd_a);
    hipFree(dd_b);
    hipFree(dd_c);
    hipFree(dd_y);
}

__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < thread_count)
    {
        int idx = ((1 << (depth + 1)) - 1) + (i * (1 << (depth + 1)));
        int offset = 1 << depth;
        //printf("f: i:%d, idx:%d, a:%f, b:%f, c:%f, y:%f,\n", i, idx, a[idx], b[idx], c[idx], y[idx]);
        if (idx < size)
        {
            if (idx + offset > size - 1)
            {
                double alpha = -a[idx] / b[idx - offset];
                a[idx] = alpha * a[idx - offset];
                b[idx] = b[idx] + (alpha * c[idx - offset]);
                y[idx] = y[idx] + (alpha * y[idx - offset]);
            }
            else
            {
                double alpha = -a[idx] / b[idx - offset];
                double beta = -c[idx] / b[idx + offset];
                a[idx] = alpha * a[idx - offset];
                b[idx] = b[idx] + alpha * c[idx - offset] + beta * a[idx + offset];
                c[idx] = beta * c[idx + offset];
                y[idx] = y[idx] + alpha * y[idx - offset] + beta * y[idx + offset];
            }
        }
    }
}

__global__ void elimination(double* a, double* b, double* c, double* x, double* y, int depth, int size_padded, int size)
{
    int id_first = ((1 << depth) - 1);
    int id_second = ((1 << depth) - 1) + (1 << depth);
    y[id_first] = y[id_first] / b[id_first];
    c[id_first] = c[id_first] / b[id_first];
    if (size == size_padded) {
        x[id_second] = (y[id_second] - (a[id_second] * y[id_first])) / (b[id_second] - (a[id_second] * c[id_first]));
        x[id_first] = y[id_first] - (c[id_first] * x[id_second]);
    }
    else {
        x[id_first] = y[id_first] - c[id_first];
    }
}

__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < thread_count)
    {
        int idx = ((1 << depth) - 1) + i * (1 << (depth + 1));
        int offset = (1 << depth);
        if (idx < size)
        {
            if (idx + offset > size - 1)
            {
                x[idx] = (y[idx] - a[idx] * x[idx - offset]) / b[idx];
            }
            else {
                x[idx] = (y[idx] - a[idx] * x[idx - offset] - c[idx] * x[idx + offset]) / b[idx];
            }
        }
        //printf("x:%f,\n", x[idx]);
    }
}


void print_arr_3d(double arr[], int nx, int ny, int nz)
{
    for (int z = 0; z < nz; z++)
    {
        for (int y = 0; y < ny; y++)
        {
            for (int x = 0; x < nx; x++)
            {
                std::cout << arr[z * ny * nx + y * nx + x] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "--------------------------" << std::endl;
    }
}

int loadImage(const char* filename, int real_width, int real_height, int* mask) {
    int width, height, channels;
    bool resize = false;

    unsigned char* input = stbi_load(filename, &width, &height, &channels, 1);
    if (!input) {
        std::cerr << "Failed to load image\n";
        return 1;
    }

    std::vector<unsigned char> resized(real_width * real_height);
    if (width != real_width || height != real_height) {
        resize = true;
        std::cout << "IMAGE SIZE DIFFERENT. RESIZING" << std::endl;
        int success = stbir_resize_uint8(
            input, width, height, 0,
            resized.data(), real_width, real_height, 0,
            1
        );
        if (!success) {
            std::cerr << "Resize failed\n";
            return 1;
        }
        stbi_image_free(input);
    }

    for (int y = 0; y < real_height; ++y) {
        for (int x = 0; x < real_width; ++x) {
            int i = y * real_width + x;
            if (resize) {
                mask[i] = resized[i] < 128 ? 0 : 1;
            }
            else {
                mask[i] = input[i] < 128 ? 0 : 1;
            }
        }
    }

    return 0;
}

int readCSVFlattened(const std::string& filename, int* mask, int width, int height) {
    std::vector<int> data;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "Failed to open file\n";
        return 1;
    }

    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string cell;

        while (std::getline(ss, cell, ',')) {
            data.push_back(std::stoi(cell));
        }
    }

    for (size_t i = 0; i < width * height; ++i) {
        mask[i] = data[i];
    }

    return 0;
}

int check_double(char* in, double* out) {
    if (in == NULL) { return 1; }
    char* end;
    double value = strtod(in, &end);

    if (*end == '\0') {
        *out = value;
        return 0;
    }
    else {
        fprintf(stderr, "Invalid input: %s\n", end);
        return 1;
    }
}

int check_and_apply(double* output, char* D, char* T, char* dt, char* l, char* n, char* c0) {
    if (check_double(D, &output[0]) == 1) { return 1; }
    if (check_double(T, &output[1]) == 1) { return 1; }
    if (check_double(dt, &output[2]) == 1) { return 1; }
    if (check_double(l, &output[3]) == 1) { return 1; }
    if (check_double(n, &output[4]) == 1) { return 1; }
    if (check_double(c0, &output[5]) == 1) { return 1; }
    return 0;
}

int check_and_apply(double* output, char* D, char* T, char* dt, char* lx, char* ly, char* lz, char* nx, char* ny, char* nz, char* c0)
{
    if (check_double(D, &output[0]) == 1) { return 1; }
    if (check_double(T, &output[1]) == 1) { return 1; }
    if (check_double(dt, &output[2]) == 1) { return 1; }
    if (check_double(lx, &output[3]) == 1) { return 1; }
    if (check_double(ly, &output[4]) == 1) { return 1; }
    if (check_double(lz, &output[5]) == 1) { return 1; }
    if (check_double(nx, &output[6]) == 1) { return 1; }
    if (check_double(ny, &output[7]) == 1) { return 1; }
    if (check_double(nz, &output[8]) == 1) { return 1; }
    if (check_double(c0, &output[9]) == 1) { return 1; }
    return 0;
}

void print_progress_bar(int i, int nt) {
    printf("\r%d/%d", i, nt);  // Move to beginning of line
    fflush(stdout);  // Force output flush
}