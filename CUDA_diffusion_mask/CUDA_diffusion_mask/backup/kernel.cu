#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <chrono>
#include <vector>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb_image_resize.h"

#define FREE_MEMORY \
    do { \
        return 0; \
    } while(0);

#define CUDA_CALL_AND_CHECK(call, msg) \
    do { \
        cuda_error = call; \
        if (cuda_error != hipSuccess) { \
            printf("Example FAILED: CUDA API returned error = %d, details: " #msg "\n", cuda_error); \
            FREE_MEMORY; \
            return -1; \
        } \
    } while(0);

void print_arr(double arr[], int size);
void print_arr(int arr[], int size);
void print_arr_3d(double arr[], int n);
void write_to_csv(double arr[], int n, const char* filename);
void print_debug(double* a, double* b, double* c, double* x, double* y, int size);

void setAx(double Ax_a[], double Ax_b[], double Ax_c[], int n, double sigma);
void setAy(double Ay_a[], double Ay_b[], double Ay_c[], int mask[], int n, double sigma);
void setAz(double Az_a[], double Az_b[], double Az_c[], int n, double sigma);

void fixEdges(double C[], int n);

__global__ void generate_bx(double* C, double* bx, int n);
__global__ void generate_by(double* C, double* Cx, double* by, int* mask, int n);
__global__ void generate_bz(double* C, double* Cy, double* bz, int n);
__global__ void update_C(double* C, double* Cz, int* mask, int n);

void solve_cr(double* d_a, double* d_b, double* d_c, double* d_x, double* d_y, int size);
__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size);
__global__ void elimination(double* a, double* b, double* c, double* x, double* y, int depth, int size_padded, int size);
__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size);

int loadImage(const char* filename, int real_width, int real_height, int* mask);
int readCSVFlattened(const std::string & filename, int* mask, int width, int height);
int check_double(char* in, double* out);
int check_and_apply(double* output, char* D, char* T, char* dt, char* l, char* n, char* c0);
void print_progress_bar(int i, int nt);

const int block_size = 512;
__device__ double device_constant[2];

hipError_t cuda_error = hipSuccess;

int main(int argc, char* argv[])
{
    if (argc != 9) {
        return 1;
    }

    // process constants
    double output[6];
    check_and_apply(output, argv[3], argv[4], argv[5], argv[6], argv[7],
        argv[8]);

    double D = output[0];
    double T = output[1];
    double dt = output[2];
    double l = output[3];
    int n = (int)output[4];
    double c0 = output[5];

    printf("Parameter:\nD:%f\nT:%f\ndt:%f\nl:%f\nn:%d\nc0:%f\n", D, T, dt, l, n, c0);

    // process file input
    const char* filename;
    filename = argv[1];

    const char* dot = strrchr(filename, '.');
    printf("File type: %s\n", dot);

    // initialize mask array
    int real_width = (n - 2);
    int real_height = (n - 2);
    int* mask = new int[real_height * real_width];

    if (strcmp(dot + 1, "csv") == 0) {
        int err = readCSVFlattened(filename, mask, real_width, real_height);
        if (err == 1) {
            return err;
        }
    }
    else if (strcmp(dot + 1, "png") == 0) {
        int err = loadImage(filename, real_width, real_height, mask);
        if (err == 1) {
            return err;
        }
    }
    else {
        return 1;
    }

    const char* output_name = strcat(argv[2], ".csv");
    printf("Output: %s\n", output_name);

    // calculate parameter needed for simulation
    const int nt = int(T / dt);
    const double d = l / (n - 1);
    const double sigma = D * dt / pow(d, 2);

    const int size = (n - 2) * (n - 2) * (n - 2); // matrix size harus 2^n

    // move constants to device
    double h_array[2] = { sigma, c0 };
    hipMemcpyToSymbol(HIP_SYMBOL(device_constant), h_array, sizeof(h_array));

    // initial condition
    double* C = (double*)calloc(n * n * n, sizeof(double));

    for (int z = 1; z < n - 1; z++)
    {
        for (int x = 1; x < n - 1; x++)
        {
            int y = n - 1;
            int idx = z * n * n + y * n + x;
            int id_mask = (z - 1) * (n - 2) + (x - 1);
            C[idx] = c0 * (1 - mask[id_mask]);
        }
    }

    // allocate memory on device
    double* Ax_a = (double*)calloc(size, sizeof(double));
    double* Ax_b = (double*)calloc(size, sizeof(double));
    double* Ax_c = (double*)calloc(size, sizeof(double));

    double* Ay_a = (double*)calloc(size, sizeof(double));
    double* Ay_b = (double*)calloc(size, sizeof(double));
    double* Ay_c = (double*)calloc(size, sizeof(double));

    double* Az_a = (double*)calloc(size, sizeof(double));
    double* Az_b = (double*)calloc(size, sizeof(double));
    double* Az_c = (double*)calloc(size, sizeof(double));

    // initialize Ax, Ay, Az
    setAx(Ax_a, Ax_b, Ax_c, n, sigma);
    setAy(Ay_a, Ay_b, Ay_c, mask, n, sigma);
    setAz(Az_a, Az_b, Az_c, n, sigma);

    double* d_Ax_a, * d_Ax_b, * d_Ax_c;
    double* d_Ay_a, * d_Ay_b, * d_Ay_c;
    double* d_Az_a, * d_Az_b, * d_Az_c;
    double* d_bx, * d_by, * d_bz;
    double* d_Cx, * d_Cy, * d_Cz, * d_C;
    int* d_mask;

    //allocate memory on device
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ax_a, (size) * sizeof(double)), "d_Ax_a");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ax_b, (size) * sizeof(double)), "d_Ax_b");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ax_c, (size) * sizeof(double)), "d_Ax_c");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ay_a, (size) * sizeof(double)), "d_Ay_a");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ay_b, (size) * sizeof(double)), "d_Ay_b");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Ay_c, (size) * sizeof(double)), "d_Ay_c");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Az_a, (size) * sizeof(double)), "d_Az_a");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Az_b, (size) * sizeof(double)), "d_Az_b");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Az_c, (size) * sizeof(double)), "d_Az_c");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_bx, (size) * sizeof(double)), "d_bx");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_by, (size) * sizeof(double)), "d_by");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_bz, (size) * sizeof(double)), "d_bz");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Cx, (size) * sizeof(double)), "d_Cx");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Cy, (size) * sizeof(double)), "d_Cy");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_Cz, (size) * sizeof(double)), "d_Cz");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_C, (n * n * n) * sizeof(double)), "d_C");
    CUDA_CALL_AND_CHECK(hipMalloc(&d_mask, (n - 2) * (n - 2) * sizeof(int)), "d_mask");

    // copy host memory to device
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ax_a, Ax_a, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ax_b, Ax_b, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ax_c, Ax_c, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ay_a, Ay_a, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ay_b, Ay_b, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Ay_c, Ay_c, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Az_a, Az_a, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Az_b, Az_b, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_Az_c, Az_c, (size) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_C, C, (n * n * n) * sizeof(double), hipMemcpyHostToDevice), "");
    CUDA_CALL_AND_CHECK(hipMemcpy(d_mask, mask, (n - 2) * (n - 2) * sizeof(int), hipMemcpyHostToDevice), "");

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    printf("SIMULATING...\n");
    // start simulation
    for (int i = 0; i < nt; i++)
    {
        int block_count = (int)ceil(float(size) / block_size);

        generate_bx<<<block_count, block_size>>>(d_C, d_bx, n);
        //print_debug(d_Ax_a, d_Ax_b, d_Ax_c, d_Cx, d_bx, size);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for C'
        solve_cr(d_Ax_a, d_Ax_b, d_Ax_c, d_Cx, d_bx, size);

        generate_by<<<block_count, block_size>>>(d_C, d_Cx, d_by, d_mask, n);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for C''
        solve_cr(d_Ay_a, d_Ay_b, d_Ay_c, d_Cy, d_by, size);

        generate_bz<<<block_count, block_size>>>(d_C, d_Cy, d_bz, n);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for Ct+1
        solve_cr(d_Az_a, d_Az_b, d_Az_c, d_Cz, d_bz, size);

        update_C<<<block_count, block_size>>>(d_C, d_Cz, d_mask, n);
        hipDeviceSynchronize();

        print_progress_bar(i, nt);
    }
    printf("\n");
    CUDA_CALL_AND_CHECK(hipMemcpy(C, d_C, (n * n * n) * sizeof(double), hipMemcpyDeviceToHost), "");

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;
    
    fixEdges(C, n);
    //print_arr_3d(C, n);
    write_to_csv(C, n, output_name);
    return 0;
}

void print_arr(double arr[], int size)
{
    printf("[");
    for (int i = 0; i < size - 1; i++)
    {
        printf("%f, ", arr[i]);
    }
    printf("%f", arr[size - 1]);
    printf("]\n");
}

void print_arr(int arr[], int size)
{
    printf("[");
    for (int i = 0; i < size - 1; i++)
    {
        printf("%d, ", arr[i]);
    }
    printf("%d", arr[size - 1]);
    printf("]\n");
}

void write_to_csv(double arr[], int n, const char* filename) {
    std::ofstream myfile;
    myfile.open(filename);

    for (int z = 0; z < n; z++)
    {
        for (int y = 0; y < n; y++)
        {
            for (int x = 0; x < n; x++)
            {
                myfile << arr[z * n * n + y * n + x] << ",";
            }
        }
    }
    myfile << n << "," << n << "," << n;
    myfile.close();
}

void print_debug(double* d_a, double* d_b, double* d_c, double* d_x, double* d_y, int size)
{
    double* a = new double[size];
    double* b = new double[size];
    double* c = new double[size];
    double* x = new double[size];
    double* y = new double[size];

    hipMemcpy(a, d_a, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(x, d_x, (size) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, (size) * sizeof(double), hipMemcpyDeviceToHost);

    printf("----------------------\n");
    print_arr(a, size);
    print_arr(b, size);
    print_arr(c, size);
    print_arr(x, size);
    print_arr(y, size);
    printf("----------------------\n");

    delete[]a;
    delete[]b;
    delete[]c;
    delete[]x;
    delete[]y;
}

void setAx(double Ax_a[], double Ax_b[], double Ax_c[], int n, double sigma)
{
    for (int z = 0; z < n - 2; z++)
    {
        for (int y = 0; y < n - 2; y++)
        {
            for (int x = 0; x < n - 2; x++)
            {
                int idx = z * (n - 2) * (n - 2) + y * (n - 2) + x;

                if (x == 0) {
                    Ax_c[idx] = -0.5 * sigma;
                    Ax_b[idx] = 1 + (0.5 * sigma);
                }
                else if (x == n - 3) {
                    Ax_a[idx] = -0.5 * sigma;
                    Ax_b[idx] = 1 + (0.5 * sigma);
                }
                else {
                    Ax_c[idx] = -0.5 * sigma;
                    Ax_b[idx] = (1.0 + sigma);
                    Ax_a[idx] = -0.5 * sigma;
                }
            }
        }
    }
}

void setAy(double Ay_a[], double Ay_b[], double Ay_c[], int mask[], int n, double sigma)
{
    for (int z = 0; z < n - 2; z++)
    {
        for (int y = 0; y < n - 2; y++)
        {
            for (int x = 0; x < n - 2; x++)
            {
                int idy = z * (n - 2) * (n - 2) + x * (n - 2) + y;
                int id_mask = z * (n - 2) + x;

                if (y == 0) {
                    Ay_c[idy] = -0.5 * sigma;
                    Ay_b[idy] = 1.0 + (0.5 * sigma);
                }
                else if (y == n - 3) {
                    Ay_a[idy] = -0.5 * sigma;
                    if (mask[id_mask] == 1) {
                        Ay_b[idy] = 1 + (0.5 * sigma);
                    }
                    else {
                        Ay_b[idy] = (1.0 + sigma);
                    }
                }
                else {
                    Ay_c[idy] = -0.5 * sigma;
                    Ay_b[idy] = (1.0 + sigma);
                    Ay_a[idy] = -0.5 * sigma;
                }
            }
        }
    }
}

void setAz(double Az_a[], double Az_b[], double Az_c[], int n, double sigma)
{
    for (int z = 0; z < n - 2; z++)
    {
        for (int y = 0; y < n - 2; y++)
        {
            for (int x = 0; x < n - 2; x++)
            {
                int idz = x * (n - 2) * (n - 2) + y * (n - 2) + z;

                if (z == 0) {
                    Az_b[idz] = 1 + (0.5 * sigma);
                    Az_c[idz] = -0.5 * sigma;
                }
                else if (z == n - 3) {
                    Az_a[idz] = -0.5 * sigma;
                    Az_b[idz] = 1 + (0.5 * sigma);
                }
                else {
                    Az_a[idz] = -0.5 * sigma;
                    Az_b[idz] = (1.0 + sigma);
                    Az_c[idz] = -0.5 * sigma;
                }
            }
        }
    }
}

void fixEdges(double C[], int n)
{
    for (int i = 0; i < n; i++) {
        C[(n - 1) * n + i] = C[n * n + (n - 1) * n + i];
        C[(n - 1) * n * n + (n - 1) * n + i] = C[(n - 2) * n * n + (n - 1) * n + i];
        C[i * n * n + (n - 1) * n] = C[i * n * n + (n - 1) * n + 1];
        C[i * n * n + (n - 1) * n + (n - 1)] = C[i * n * n + (n - 1) * n + (n - 2)];

        C[i] = C[n * n + i];
        C[(n - 1) * n * n + i] = C[(n - 2) * n * n + i];
        C[i * n * n] = C[i * n * n + 1];
        C[i * n * n + (n - 1)] = C[i * n * n + (n - 2)];

        C[i * n] = C[i * n + 1];
        C[i * n + (n - 1)] = C[i * n + (n - 2)];
        C[(n - 1) * n * n + i * n] = C[(n - 1) * n * n + i * n + 1];
        C[(n - 1) * n * n + i * n + (n - 1)] = C[(n - 1) * n * n + i * n + (n - 2)];
    }
}

__global__ void generate_bx(double* C, double* bx, int n)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (n - 2) * (n - 2) * (n - 2))
    {
        double sigma = device_constant[0];

        int x = i % (n - 2);
        int y = (i / (n - 2)) % (n - 2);
        int z = i / ((n - 2) * (n - 2));

        int idx = i;
        int c_id = (z + 1) * n * n + (y + 1) * n + (x + 1);

        int left = c_id - 1;
        int right = c_id + 1;
        int up = c_id + n;
        int down = c_id - n;
        int front = c_id + n * n;
        int back = c_id - n * n;

        bx[idx] = C[c_id] + 0.5 * sigma * (C[left] - 2 * C[c_id] + C[right])
            + sigma * (C[down] - 2 * C[c_id] + C[up])
            + sigma * (C[back] - 2 * C[c_id] + C[front]);
    }
}

__global__ void generate_by(double* C, double* Cx, double* by, int* mask, int n)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (n - 2) * (n - 2) * (n - 2))
    {
        double sigma = device_constant[0];
        double c0 = device_constant[1];

        int x = i % (n - 2);
        int y = (i / (n - 2)) % (n - 2);
        int z = i / ((n - 2) * (n - 2));

        int idx = i;
        int idy = z * (n - 2) * (n - 2) + x * (n - 2) + y;
        int id_mask = z * (n - 2) + x;
        int c_id = (z + 1) * n * n + (y + 1) * n + (x + 1);

        int up = c_id + n;
        int down = c_id - n;

        by[idy] = Cx[idx] - 0.5 * sigma * (C[down] - 2 * C[c_id] + C[up]);

        if (y == n - 3)
        {
            if (mask[id_mask] == 0)
            {
                by[idy] += c0 * (0.5 * sigma);
            }
        }
    }
}

__global__ void generate_bz(double* C, double* Cy, double* bz, int n)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (n - 2) * (n - 2) * (n - 2))
    {
        double sigma = device_constant[0];

        int x = i % (n - 2);
        int y = (i / (n - 2)) % (n - 2);
        int z = i / ((n - 2) * (n - 2));

        int idy = z * (n - 2) * (n - 2) + x * (n - 2) + y;
        int idz = x * (n - 2) * (n - 2) + y * (n - 2) + z;
        int c_id = (z + 1) * n * n + (y + 1) * n + (x + 1);

        int front = c_id + n * n;
        int back = c_id - n * n;

        bz[idz] = Cy[idy] - 0.5 * sigma * (C[back] - 2 * C[c_id] + C[front]);
    }
}

__global__ void update_C(double* C, double* Cz, int* mask, int n)
{
    int i = blockIdx.x * block_size + threadIdx.x;

    if (i < (n - 2) * (n - 2) * (n - 2))
    {
        int x = i % (n - 2);
        int y = (i / (n - 2)) % (n - 2);
        int z = i / ((n - 2) * (n - 2));

        int idz = x * (n - 2) * (n - 2) + y * (n - 2) + z;
        int id_mask = z * (n - 2) + x;
        int c_id = (z + 1) * n * n + (y + 1) * n + (x + 1);

        C[c_id] = Cz[idz];

        int left = c_id - 1;
        int right = c_id + 1;
        int up = c_id + n;
        int down = c_id - n;
        int front = c_id + n * n;
        int back = c_id - n * n;

        if (x == 0) {
            C[left] = C[c_id];
        }
        if (x == n - 3) {
            C[right] = C[c_id];
        }
        if (y == 0) {
            C[down] = C[c_id];
        }
        if (y == n - 3) {
            if (mask[id_mask] == 1) {
                C[up] = C[c_id];
            }
        }
        if (z == 0) {
            C[back] = C[c_id];
        }
        if (z == n - 3) {
            C[front] = C[c_id];
        }
    }
}

// input is array at global memory, output d_x
void solve_cr(double* d_a, double* d_b, double* d_c, double* d_x, double* d_y, int size)
{
    int size_padded = 1 << (int)(ceil(log2(size)));
    int depth = int(log2(size_padded)) - 1;

    // handle malloc at global memory
    double* dd_a, * dd_b, * dd_c, * dd_y;
    hipMalloc(&dd_a, (size) * sizeof(double));
    hipMalloc(&dd_b, (size) * sizeof(double));
    hipMalloc(&dd_c, (size) * sizeof(double));
    hipMalloc(&dd_y, (size) * sizeof(double));

    hipMemcpy(dd_a, d_a, (size) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_b, d_b, (size) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_c, d_c, (size) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_y, d_y, (size) * sizeof(double), hipMemcpyDeviceToDevice);

    // forward reduction
    for (int i = 0; i < depth; i++)
    {
        int thread_count = size_padded / (1 << (i + 1));
        int block_count = (thread_count + block_size - 1) / block_size;

        forward_reduce << <block_count, block_size >> > (dd_a, dd_b, dd_c, d_x, dd_y, i, thread_count, size);
        hipDeviceSynchronize();
    }

    // elimination
    elimination << <1, 1 >> > (dd_a, dd_b, dd_c, d_x, dd_y, depth, size_padded, size);
    hipDeviceSynchronize();

    // backward subtitution
    for (int i = depth - 1; i >= 0; i--)
    {
        int thread_count = size_padded / (1 << (i + 1));
        int block_count = (thread_count + block_size - 1) / block_size;

        backward_subs << <block_count, block_size >> > (dd_a, dd_b, dd_c, d_x, dd_y, i, thread_count, size);
        hipDeviceSynchronize();
    }

    hipFree(dd_a);
    hipFree(dd_b);
    hipFree(dd_c);
    hipFree(dd_y);
}

__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < thread_count)
    {
        int idx = ((1 << (depth + 1)) - 1) + (i * (1 << (depth + 1)));
        int offset = 1 << depth;
        //printf("f: i:%d, idx:%d, a:%f, b:%f, c:%f, y:%f,\n", i, idx, a[idx], b[idx], c[idx], y[idx]);
        if (idx < size)
        {
            if (idx + offset > size - 1)
            {
                double alpha = -a[idx] / b[idx - offset];
                a[idx] = alpha * a[idx - offset];
                b[idx] = b[idx] + (alpha * c[idx - offset]);
                y[idx] = y[idx] + (alpha * y[idx - offset]);
            }
            else
            {
                double alpha = -a[idx] / b[idx - offset];
                double beta = -c[idx] / b[idx + offset];
                a[idx] = alpha * a[idx - offset];
                b[idx] = b[idx] + alpha * c[idx - offset] + beta * a[idx + offset];
                c[idx] = beta * c[idx + offset];
                y[idx] = y[idx] + alpha * y[idx - offset] + beta * y[idx + offset];
            }
        }
    }
}

__global__ void elimination(double* a, double* b, double* c, double* x, double* y, int depth, int size_padded, int size)
{
    int id_first = ((1 << depth) - 1);
    int id_second = ((1 << depth) - 1) + (1 << depth);
    y[id_first] = y[id_first] / b[id_first];
    c[id_first] = c[id_first] / b[id_first];
    if (size == size_padded) {
        x[id_second] = (y[id_second] - (a[id_second] * y[id_first])) / (b[id_second] - (a[id_second] * c[id_first]));
        x[id_first] = y[id_first] - (c[id_first] * x[id_second]);
    }
    else {
        x[id_first] = y[id_first] - c[id_first];
    }
}

__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < thread_count)
    {
        int idx = ((1 << depth) - 1) + i * (1 << (depth + 1));
        int offset = (1 << depth);
        if (idx < size)
        {
            if (idx + offset > size - 1)
            {
                x[idx] = (y[idx] - a[idx] * x[idx - offset]) / b[idx];
            }
            else {
                x[idx] = (y[idx] - a[idx] * x[idx - offset] - c[idx] * x[idx + offset]) / b[idx];
            }
        }
        //printf("x:%f,\n", x[idx]);
    }
}


void print_arr_3d(double arr[], int n)
{
    for (int z = 0; z < n; z++)
    {
        for (int y = 0; y < n; y++)
        {
            for (int x = 0; x < n; x++)
            {
                std::cout << arr[z * n * n + y * n + x] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "--------------------------" << std::endl;
    }
}

int loadImage(const char* filename, int real_width, int real_height, int* mask) {
    int width, height, channels;
    bool resize = false;

    unsigned char* input = stbi_load(filename, &width, &height, &channels, 1);
    if (!input) {
        std::cerr << "Failed to load image\n";
        return 1;
    }

    std::vector<unsigned char> resized(real_width * real_height);
    if (width != real_width || height != real_height) {
        resize = true;
        std::cout << "IMAGE SIZE DIFFERENT. RESIZING" << std::endl;
        int success = stbir_resize_uint8(
            input, width, height, 0,
            resized.data(), real_width, real_height, 0,
            1
        );
        if (!success) {
            std::cerr << "Resize failed\n";
            return 1;
        }
        stbi_image_free(input);
    }

    for (int y = 0; y < real_height; ++y) {
        for (int x = 0; x < real_width; ++x) {
            int i = y * real_width + x;
            if (resize) {
                mask[i] = resized[i] < 128 ? 0 : 1;
            }
            else {
                mask[i] = input[i] < 128 ? 0 : 1;
            }
        }
    }

    return 0;
}

int readCSVFlattened(const std::string& filename, int* mask, int width, int height) {
    std::vector<int> data;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "Failed to open file\n";
        return 1;
    }

    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string cell;

        while (std::getline(ss, cell, ',')) {
            data.push_back(std::stoi(cell));
        }
    }

    for (size_t i = 0; i < width * height; ++i) {
        mask[i] = data[i];
    }

    return 0;
}

int check_double(char* in, double* out) {
    if (in == NULL) { return 1; }
    char* end;
    double value = strtod(in, &end);

    if (*end == '\0') {
        *out = value;
        return 0;
    }
    else {
        printf("Invalid input: %s\n", end);
        return 1;
    }
}

int check_and_apply(double* output, char* D, char* T, char* dt, char* l, char* n, char* c0) {
    if (check_double(D, &output[0]) == 1) { return 1; }
    if (check_double(T, &output[1]) == 1) { return 1; }
    if (check_double(dt, &output[2]) == 1) { return 1; }
    if (check_double(l, &output[3]) == 1) { return 1; }
    if (check_double(n, &output[4]) == 1) { return 1; }
    if (check_double(c0, &output[5]) == 1) { return 1; }
    return 0;
}

void print_progress_bar(int i, int nt) {
    printf("\r%d/%d", i, nt);  // Move to beginning of line
    fflush(stdout);  // Force output flush
}