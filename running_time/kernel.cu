#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

#define CUDA_CALL_AND_CHECK(call, msg) \
    do { \
        cuda_error = call; \
        if (cuda_error != hipSuccess) { \
            printf("Example FAILED: CUDA API returned error = %d, details: " #msg "\n", cuda_error); \
            return -1; \
        } \
    } while(0);

const int size = 65536; // matrix size harus 2^n
const int block_size = 32;
long total_time = 0;

void print_arr(double arr[], int size);
void generate_diag(double arr[], int size, int val);
void generate_arr(double arr[], int size);
hipError_t solve_cr(double* a, double* b, double* c, double* x, double* y);
__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth);
__global__ void elimination(double* a, double* b, double* c, double* x, double* y);
__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth);

int main()
{
    hipError_t cuda_error = hipSuccess;

    int run_times = 100;

    for (int i = 0; i < run_times + 1; i++)
    {
        double* h_a = new double[size - 1];
        double* h_b = new double[size];
        double* h_c = new double[size - 1];
        double* h_x = new double[size];
        double* h_y = new double[size];

        generate_diag(h_a, size - 1, 1);
        generate_diag(h_b, size, 2);
        generate_diag(h_c, size - 1, 1);
        generate_diag(h_x, size, 0);
        generate_diag(h_y, size, 1);

        // solve tridiagonal matrix using cyclic reduction method
        CUDA_CALL_AND_CHECK(solve_cr(h_a, h_b, h_c, h_x, h_y), "");

        delete[]h_a;
        delete[]h_b;
        delete[]h_c;
        delete[]h_x;
        delete[]h_y;

        // ignore first run
        if (i == 0) { total_time = 0; }
    }

    std::cout << "Avg time = " << (double)total_time / (double)run_times << std::endl;

    return 0;
}

void generate_diag(double arr[], int size, int val)
{
    for (int i = 0; i < size; i++)
    {
        arr[i] = val;
    }
}


void generate_arr(double arr[], int size)
{
    for (int i = 0; i < size; i++) 
    {
        arr[i] = (rand() % 100);
    }
}

void print_arr(double arr[], int size)
{
    printf("[");
    for (int i=0; i < size-1; i++) 
    {
        printf("%.2f, ", arr[i]);
    }
    printf("%.2f", arr[size-1]);
    printf("]\n");
}

hipError_t solve_cr(double* a, double* b, double* c, double* x, double* y)
{
    hipError_t cuda_error = hipSuccess;
    int depth = int(log2(size)) - 1;

    // handle malloc at global memory
    double* d_a, * d_b, * d_c, * d_x, * d_y;
    cuda_error = hipMalloc(&d_a, (size - 1) * sizeof(double));
    if (cuda_error != hipSuccess) { 
        goto Error;
    }
    cuda_error = hipMalloc(&d_b, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMalloc(&d_c, (size - 1) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMalloc(&d_x, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMalloc(&d_y, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }

    cuda_error = hipMemcpy(d_a, a, (size - 1) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_b, b, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_c, c, (size - 1) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_x, x, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_y, y, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }

    // start timer
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    // forward reduction
    for (int i = 0; i < depth; i++)
    {
        int thread_count = (size / pow(2, i + 1));
        int block_count = 1;
        if (thread_count > block_size) {
            block_count = ceil(thread_count / block_size);
            thread_count = block_size;
        }
        forward_reduce << <block_count, thread_count >> > (d_a, d_b, d_c, d_x, d_y, i);
        cuda_error = hipGetLastError();
        if (cuda_error != hipSuccess) {
            fprintf(stderr, "forward_reduce launch failed: %s\n", hipGetErrorString(cuda_error));
            goto Error;
        }

        cuda_error = hipDeviceSynchronize();
        if (cuda_error != hipSuccess) {
            goto Error;
        }
    }

    // elimination
    elimination << <1, 1 >> > (d_a, d_b, d_c, d_x, d_y);
    cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "elimination launch failed: %s\n", hipGetErrorString(cuda_error));
        goto Error;
    }
    cuda_error = hipDeviceSynchronize();
    if (cuda_error != hipSuccess) {
        goto Error;
    }

    // backward subtitution
    for (int i = depth - 1; i >= 0; i--)
    {
        int thread_count = (size / pow(2, i + 1));
        int block_count = 1;
        if (thread_count > block_size) {
            block_count = ceil(thread_count / block_size);
            thread_count = block_size;
        }

        backward_subs << <block_count, thread_count >> > (d_a, d_b, d_c, d_x, d_y, i);
        cuda_error = hipGetLastError();
        if (cuda_error != hipSuccess) {
            fprintf(stderr, "backward_subs launch failed: %s\n", hipGetErrorString(cuda_error));
            goto Error;
        }
        cuda_error = hipDeviceSynchronize();
        if (cuda_error != hipSuccess) {
            goto Error;
        }
    }

    // record time
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    long time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    std::cout << "Running Time = " << time << "[microsec]" << std::endl;
    total_time += time;

    cuda_error = hipMemcpy(a, d_a, (size - 1) * sizeof(double), hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(b, d_b, (size) * sizeof(double), hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(c, d_c, (size - 1) * sizeof(double), hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(x, d_x, (size) * sizeof(double), hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(y, d_y, (size) * sizeof(double), hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        goto Error;
    }

Error:
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_x);
    hipFree(d_y);
    return cuda_error;
}

__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < size)
    {
        int idx = ((1 << depth + 1) - 1) + (i * (1 << depth + 1));
        int offset = 1 << depth;

        if (idx == (1 << depth + 1) - 1)
        {
            double alpha = -a[idx - 1] / b[idx - offset];
            double beta = -c[idx] / b[idx + offset];
            a[idx - 1] = 0;
            b[idx] = b[idx] + alpha * c[idx - offset] + beta * a[idx + offset - 1];
            c[idx] = beta * c[idx + offset];
            y[idx] = y[idx] + alpha * y[idx - offset] + beta * y[idx + offset];
        }
        else if (idx == size - 1)
        {
            double alpha = -a[idx - 1] / b[idx - offset];
            a[idx - 1] = alpha * a[idx - offset - 1];
            b[idx] = b[idx] + alpha * c[idx - offset];
            y[idx] = y[idx] + alpha * y[idx - offset];
        }
        else
        {
            double alpha = -a[idx - 1] / b[idx - offset];
            double beta = -c[idx] / b[idx + offset];
            a[idx - 1] = alpha * a[idx - offset - 1];
            b[idx] = b[idx] + alpha * c[idx - offset] + beta * a[idx + offset - 1];
            c[idx] = beta * c[idx + offset];
            y[idx] = y[idx] + alpha * y[idx - offset] + beta * y[idx + offset];
        }
    }
}

__global__ void elimination(double* a, double* b, double* c, double* x, double* y)
{
    // elimination to get X(n / 2 - 1) and X(n - 1)
    int idx_mid = int(size / 2) - 1;
    y[idx_mid] = y[idx_mid] / b[idx_mid];
    c[idx_mid] = c[idx_mid] / b[idx_mid];
    x[size - 1] = (y[size - 1] - (a[size - 2] * y[idx_mid])) / (b[size - 1] - (a[size - 2] * c[idx_mid]));
    x[idx_mid] = y[idx_mid] - c[idx_mid] * x[size - 1];
}

__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < size)
    {
        if (depth == 0)
        {
            int idx = i * 2;
            if (idx == 0)
            {
                x[idx] = (y[idx] - c[idx] * x[idx + 1]) / b[idx];
            }
            else
            {
                x[idx] = (y[idx] - a[idx - 1] * x[idx - 1] - c[idx] * x[idx + 1]) / b[idx];
            }
        }
        else
        {
            int idx = ((1 << depth) - 1) + i * (1 << depth + 1);
            int offset = (1 << depth);

            x[idx] = (y[idx] - a[idx - 1] * x[idx - offset] - c[idx] * x[idx + offset]) / b[idx];
        }
    }
}
