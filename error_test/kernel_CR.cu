#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <chrono>

#define CUDA_CALL_AND_CHECK(call, msg) \
    do { \
        cuda_error = call; \
        if (cuda_error != hipSuccess) { \
            printf("Example FAILED: CUDA API returned error = %d, details: " #msg "\n", cuda_error); \
            return -1; \
        } \
    } while(0);

const int size = 10; // matrix size harus 2^n
const int block_size = 1024;
long total_time = 0;

void print_arr(double arr[], int size);
void generate_for_test(double* a, double* b, double* c, double* x, double* y, int size);
void check_error(double* a, double* b, double* c, double* x, double* y, int size);
hipError_t solve_cr(double* a, double* b, double* c, double* x, double* y);
__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size);
__global__ void elimination(double* a, double* b, double* c, double* x, double* y, int depth, int size_padded, int size);
__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size);

int main()
{
    // set random number seed
    srand(time(0));
    hipError_t cuda_error = hipSuccess;

    int run_times = 10;

    for (int i = 0; i < run_times + 1; i++)
    {
        double* h_a = new double[size];
        double* h_b = new double[size];
        double* h_c = new double[size];
        double* h_x = new double[size];
        double* h_y = new double[size];
        generate_for_test(h_a, h_b, h_c, h_x, h_y, size);

        // solve tridiagonal matrix using cyclic reduction method
        CUDA_CALL_AND_CHECK(solve_cr(h_a, h_b, h_c, h_x, h_y), "");
        check_error(h_a, h_b, h_c, h_x, h_y, size);
        print_arr(h_b, size);
        delete[]h_a;
        delete[]h_b;
        delete[]h_c;
        delete[]h_x;
        delete[]h_y;

        // ignore first run
        if (i == 0) { total_time = 0; }
    }

    std::cout << "Avg time = " << (double)total_time / (double)run_times << std::endl;

    return 0;
}


void generate_for_test(double* a, double* b, double* c, double* x, double* y, int size)
{
    for (int i = 0; i < size; i++) {
        double perturb = (rand() % 100000) / 100000.0;
        a[i] = 1.0f + perturb;
        perturb = (rand() % 100000) / 100000.0;
        b[i] = 1.0f + perturb;
        perturb = (rand() % 100000) / 100000.0;
        c[i] = 1.0f + perturb;
        perturb = (rand() % 100000) / 100000.0;
        y[i] = 1.0f + perturb;
        x[i] = 0.0f;
    }
    a[0] = 0.0f;
    c[size - 1] = 0.0f;
}

void check_error(double* a, double* b, double* c, double* x, double* y, int size)
{
    double* y_res = new double[size];
    y_res[0] = b[0] * x[0] + c[0] * x[1];
    y_res[size-1] = a[size - 1] * x[size - 2] + b[size - 1] * x[size - 1];
    for (int i = 1; i < size-1; i++) {
        y_res[i] = a[i] * x[i - 1] + b[i] * x[i] + c[i] * x[i + 1];
    }

    double total_err = 0;
    for (int i = 0; i < size; i++) {
        total_err += abs(y_res[i] - y[i]);
    }
    printf("Avg Error: %f\n", (total_err / size));
}

void print_arr(double arr[], int size)
{
    printf("[");
    for (int i = 0; i < size - 1; i++)
    {
        printf("%.2f, ", arr[i]);
    }
    printf("%.2f", arr[size - 1]);
    printf("]\n");
}

hipError_t solve_cr(double* a, double* b, double* c, double* x, double* y)
{
    hipError_t cuda_error = hipSuccess;
    int size_padded = 1 << (int)(ceil(log2(size)));
    int depth = int(log2(size_padded)) - 1;

    // handle malloc at global memory
    double* d_a, * d_b, * d_c, * d_x, * d_y;
    cuda_error = hipMalloc(&d_a, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMalloc(&d_b, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMalloc(&d_c, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMalloc(&d_x, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMalloc(&d_y, (size) * sizeof(double));
    if (cuda_error != hipSuccess) {
        goto Error;
    }

    cuda_error = hipMemcpy(d_a, a, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_b, b, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_c, c, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_x, x, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }
    cuda_error = hipMemcpy(d_y, y, (size) * sizeof(double), hipMemcpyHostToDevice);
    if (cuda_error != hipSuccess) {
        goto Error;
    }

    // start timer
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    // forward reduction
    for (int i = 0; i < depth; i++)
    {
        int thread_count = size_padded / (1 << (i + 1));
        int block_count = (thread_count + block_size - 1) / block_size;
        forward_reduce << <block_count, block_size >> > (d_a, d_b, d_c, d_x, d_y, i, thread_count, size);
        cuda_error = hipGetLastError();
        if (cuda_error != hipSuccess) {
            fprintf(stderr, "forward_reduce launch failed: %s\n", hipGetErrorString(cuda_error));
            goto Error;
        }

        cuda_error = hipDeviceSynchronize();
        if (cuda_error != hipSuccess) {
            goto Error;
        }
    }

    // elimination
    elimination << <1, 1 >> > (d_a, d_b, d_c, d_x, d_y, depth, size_padded, size);
    cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        fprintf(stderr, "elimination launch failed: %s\n", hipGetErrorString(cuda_error));
        goto Error;
    }
    cuda_error = hipDeviceSynchronize();
    if (cuda_error != hipSuccess) {
        goto Error;
    }

    // backward subtitution
    for (int i = depth - 1; i >= 0; i--)
    {
        int thread_count = size_padded / (1 << (i + 1));
        int block_count = (thread_count + block_size - 1) / block_size;

        backward_subs << <block_count, block_size >> > (d_a, d_b, d_c, d_x, d_y, i, thread_count, size);
        cuda_error = hipGetLastError();
        if (cuda_error != hipSuccess) {
            fprintf(stderr, "backward_subs launch failed: %s\n", hipGetErrorString(cuda_error));
            goto Error;
        }
        cuda_error = hipDeviceSynchronize();
        if (cuda_error != hipSuccess) {
            goto Error;
        }
    }

    // record time
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    long time = std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count();
    std::cout << "Running Time = " << time << "[microsec]" << std::endl;
    total_time += time;

    cuda_error = hipMemcpy(x, d_x, (size) * sizeof(double), hipMemcpyDeviceToHost);
    if (cuda_error != hipSuccess) {
        goto Error;
    }

Error:
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_x);
    hipFree(d_y);
    return cuda_error;
}

__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < thread_count)
    {
        int idx = ((1 << (depth + 1)) - 1) + (i * (1 << (depth + 1)));
        int offset = 1 << depth;
        //printf("f: i:%d, idx:%d, a:%f, b:%f, c:%f, y:%f,\n", i, idx, a[idx], b[idx], c[idx], y[idx]);
        if (idx < size)
        {
            if (idx + offset > size - 1)
            {
                double alpha = -a[idx] / b[idx - offset];
                a[idx] = alpha * a[idx - offset];
                b[idx] = b[idx] + (alpha * c[idx - offset]);
                y[idx] = y[idx] + (alpha * y[idx - offset]);
            }
            else
            {
                double alpha = -a[idx] / b[idx - offset];
                double beta = -c[idx] / b[idx + offset];
                a[idx] = alpha * a[idx - offset];
                b[idx] = b[idx] + alpha * c[idx - offset] + beta * a[idx + offset];
                c[idx] = beta * c[idx + offset];
                y[idx] = y[idx] + alpha * y[idx - offset] + beta * y[idx + offset];
            }
        }
    }
}

__global__ void elimination(double* a, double* b, double* c, double* x, double* y, int depth, int size_padded, int size)
{
    int id_first = ((1 << depth) - 1);
    int id_second = ((1 << depth) - 1) + (1 << depth);
    y[id_first] = y[id_first] / b[id_first];
    c[id_first] = c[id_first] / b[id_first];
    if (size == size_padded) {
        x[id_second] = (y[id_second] - (a[id_second] * y[id_first])) / (b[id_second] - (a[id_second] * c[id_first]));
        x[id_first] = y[id_first] - (c[id_first] * x[id_second]);
    }
    else {
        x[id_first] = y[id_first] - c[id_first];
    }
}

__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth, int thread_count, int size)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (i < thread_count)
    {
        int idx = ((1 << depth) - 1) + i * (1 << (depth + 1));
        int offset = (1 << depth);
        if (idx < size)
        {
            if (idx + offset > size - 1)
            {
                x[idx] = (y[idx] - a[idx] * x[idx - offset]) / b[idx];
            }
            else {
                x[idx] = (y[idx] - a[idx] * x[idx - offset] - c[idx] * x[idx + offset]) / b[idx];
            }
        }
        //printf("x:%f,\n", x[idx]);
    }
}