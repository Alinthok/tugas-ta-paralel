#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include "tridiag_solver.h"
#include <chrono>

void print_arr(double arr[], int size);
void write_to_csv(double arr[]);

__global__ void generate_bx(double* C, double* A_a, double* A_b, double* A_c, double* bx, double sigma, double c0);
__global__ void generate_by(double* C, double* Cx, double* A_a, double* A_b, double* A_c, double* by, double sigma, double c0);
__global__ void generate_bz(double* C, double* Cy, double* A_a, double* A_b, double* A_c, double* bz, double sigma, double c0);
__global__ void update_C(double* C, double* Cz);

const int n = 18;
const double dt = 0.001;
const double l = 1;
const double t = 1;
const double c0 = 1;
const double D = 0.01;

const int nt = int(t / dt);
const double d = l / (n - 1);
const double sigma = D * dt / pow(d, 2);

int main()
{
    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();

    double C[n*n*n] = {};

    for (int z = 0; z < n; z++)
    {
        for (int y = 0; y < n; y++)
        {
            for (int x = 0; x < n; x++)
            {
                if (y == 0)
                {
                    C[z * n * n + y * n + x] = c0;
                }
            }
        }
    }

    for (int z = 0; z < n; z++)
    {
        for (int y = 0; y < n; y++) 
        {
            for (int x = 0; x < n; x++) 
            {
                std::cout << C[z * n * n + y * n + x] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "--------------------------" << std::endl;
    }

    double A_a[(n - 2) * (n - 2) * (n - 2) - 1] = {};
    double A_b[(n - 2) * (n - 2) * (n - 2)] = {};
    double A_c[(n - 2) * (n - 2) * (n - 2) - 1] = {};
    
    // setup tridiagonal matrix
    for (int z = 0; z < n - 2; z++)
    {
        for (int y = 0; y < n - 2; y++) 
        {
            for (int x = 0; x < n - 2; x++) 
            {
                // A matriks sama untuk x, y, z karena hitungannya di transpose
                int id = z * (n - 2) * (n - 2) + y * (n - 2) + x;
                int left = id - 1;

                A_b[id] = 1.0 + sigma;

                if (x == 0)
                {
                    A_c[id] += -0.5 * sigma;
                }
                else if (x == n - 3)
                {
                    A_a[left] += -0.5 * sigma;
                }
                else
                {
                    A_c[id] += -0.5 * sigma;
                    A_a[left] += -0.5 * sigma;
                }
            }
        }
    }

    // allocate needed resources at global memory
    double *d_a, *d_b, *d_c, *d_bx, *d_by, *d_bz, *d_Cx, *d_Cy, *d_Cz, *d_C; 

    hipMalloc(&d_a, ((n - 2) * (n - 2) * (n - 2) - 1) * sizeof(double));
    hipMalloc(&d_b, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double));
    hipMalloc(&d_c, ((n - 2) * (n - 2) * (n - 2) - 1) * sizeof(double));
    hipMalloc(&d_bx, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double));
    hipMalloc(&d_by, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double));
    hipMalloc(&d_bz, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double));
    hipMalloc(&d_Cx, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double));
    hipMalloc(&d_Cy, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double));
    hipMalloc(&d_Cz, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double));
    hipMalloc(&d_C, (n * n * n) * sizeof(double));
    
    hipMemcpy(d_a, A_a, ((n - 2) * (n - 2) * (n - 2) - 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, A_b, ((n - 2) * (n - 2) * (n - 2)) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c, A_c, ((n - 2) * (n - 2) * (n - 2) - 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, (n * n * n) * sizeof(double), hipMemcpyHostToDevice);

    for (int i = 0; i < nt; i++)
    {
        int block_count = ceil(((n - 2) * (n - 2) * (n - 2)) / block_size);

        generate_bx<<<block_count, block_size>>>(d_C, d_a, d_b, d_c, d_bx, sigma, c0);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for C'
        solve_cr(d_a, d_b, d_c, d_Cx, d_bx);

        generate_by<<<block_count, block_size>>>(d_C, d_Cx, d_a, d_b, d_c, d_by, sigma, c0);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for C''
        solve_cr(d_a, d_b, d_c, d_Cy, d_by);

        generate_bz<<<block_count, block_size>>>(d_C, d_Cy, d_a, d_b, d_c, d_bz, sigma, c0);
        hipDeviceSynchronize();

        // solve tridiagonal matrix for Ct+1
        solve_cr(d_a, d_b, d_c, d_Cz, d_bz);

        update_C<<<block_count, block_size>>>(d_C, d_Cz);
        hipDeviceSynchronize();
    }

    hipMemcpy(C, d_C, (n * n * n) * sizeof(double), hipMemcpyDeviceToHost);

    //print_arr(A_a, (ny - 2)* (nx - 2) - 1);
    //print_arr(A_b, (ny - 2)* (nx - 2));
    //print_arr(A_c, (ny - 2)* (nx - 2) - 1);

    for (int z = 0; z < n; z++)
    {
        for (int y = 0; y < n; y++)
        {
            for (int x = 0; x < n; x++)
            {
                std::cout << C[z * n * n + y * n + x] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << "--------------------------" << std::endl;
    }

    write_to_csv(C);

    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;

    return 0;
}

void print_arr(double arr[], int size)
{
    printf("[");
    for (int i = 0; i < size - 1; i++)
    {
        printf("%.2f, ", arr[i]);
    }
    printf("%.2f", arr[size - 1]);
    printf("]\n");
}

void write_to_csv(double arr[]) {
    std::ofstream myfile;
    myfile.open("18x18x18.csv");

    for (int z = 0; z < n; z++)
    {
        for (int y = 0; y < n; y++)
        {
            for (int x = 0; x < n; x++)
            {
                myfile << arr[z * n * n + y * n + x] << ",";
            }
        }
    }

    myfile.close();
}

__global__ void generate_bx(double* C, double* A_a, double* A_b, double* A_c, double* bx, double sigma, double c0)
{
    int idx = blockIdx.x * block_size + threadIdx.x;

    if (idx < (n - 2) * (n - 2) * (n - 2))
    {
        int x = idx % (n - 2);
        int y = (idx / (n - 2)) % (n - 2);
        int z = idx / ((n - 2) * (n - 2));

        int c_id = (z + 1) * n * n + (y + 1) * n + (x + 1);

        int left = c_id - 1;
        int right = c_id + 1;
        int up = c_id + n;
        int down = c_id - n;
        int front = c_id + n * n;
        int back = c_id - n * n;

        bx[idx] = C[c_id] + 0.5 * sigma * (C[left] - 2 * C[c_id] + C[right])
            + sigma * (C[down] - 2 * C[c_id] + C[up])
            + sigma * (C[back] - 2 * C[c_id] + C[front]);

        // boundary conditions
        if (x == 0)
        {
            bx[idx] += 0 * 0.5 * sigma;
        }
        else if (x == n - 3)
        {
            bx[idx] += 0 * 0.5 * sigma;
        }
    }
}

__global__ void generate_by(double* C, double* Cx, double* A_a, double* A_b, double* A_c, double* by, double sigma, double c0)
{
    int idx = blockIdx.x * block_size + threadIdx.x;
    
    if (idx < (n - 2) * (n - 2) * (n - 2))
    {
        int x = idx % (n - 2);
        int y = (idx / (n - 2)) % (n - 2);
        int z = idx / ((n - 2) * (n - 2));

        int idt = z * (n - 2) * (n - 2) + x * (n - 2) + y;
        int c_id = (z + 1) * n * n + (y + 1) * n + (x + 1);

        int up = c_id + n;
        int down = c_id - n;

        by[idt] = Cx[idx] - 0.5 * sigma * (C[down] - 2 * C[c_id] + C[up]);

        // boundary conditions
        if (y == 0)
        {
            by[idt] += c0 * 0.5 * sigma;
        }
        else if (y == n - 3)
        {
            by[idt] += 0 * 0.5 * sigma;
        }
    }
}

__global__ void generate_bz(double* C, double* Cy, double* A_a, double* A_b, double* A_c, double* bz, double sigma, double c0)
{
    int idx = blockIdx.x * block_size + threadIdx.x;

    if (idx < (n - 2) * (n - 2) * (n - 2))
    {
        int x = idx % (n - 2);
        int y = (idx / (n - 2)) % (n - 2);
        int z = idx / ((n - 2) * (n - 2));

        int idy = z * (n - 2) * (n - 2) + x * (n - 2) + y;
        int idz = x * (n - 2) * (n - 2) + y * (n - 2) + z;
        int c_id = (z + 1) * n * n + (y + 1) * n + (x + 1);

        int front = c_id + n * n;
        int back = c_id - n * n;

        bz[idz] = Cy[idy] - 0.5 * sigma * (C[back] - 2 * C[c_id] + C[front]);

        // boundary conditions
        if (z == 0)
        {
            bz[idz] += 0 * 0.5 * sigma;
        }
        else if (z == n - 3)
        {
            bz[idz] += 0 * 0.5 * sigma;
        }
    }
}

__global__ void update_C(double* C, double* Cz)
{
    int idx = blockIdx.x * block_size + threadIdx.x;

    if (idx < (n - 2) * (n - 2) * (n - 2))
    {
        int x = idx % (n - 2);
        int y = (idx / (n - 2)) % (n - 2);
        int z = idx / ((n - 2) * (n - 2));

        int idt = x * (n - 2) * (n - 2) + y * (n - 2) + z;
        int c_id = (x + 1) * n * n + (y + 1) * n + (z + 1);

        C[c_id] = Cz[idt];
    }
}
