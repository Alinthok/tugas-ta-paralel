#include "hip/hip_runtime.h"
#include "tridiag_solver.h"

// input is array at global memory, output d_x
void solve_cr(double* d_a, double* d_b, double* d_c, double* d_x, double* d_y)
{
    int depth = int(log2(size)) - 1;

    // handle malloc at global memory
    double* dd_a, * dd_b, * dd_c, * dd_y;
    hipMalloc(&dd_a, (size - 1) * sizeof(double));
    hipMalloc(&dd_b, (size) * sizeof(double));
    hipMalloc(&dd_c, (size - 1) * sizeof(double));
    hipMalloc(&dd_y, (size) * sizeof(double));

    hipMemcpy(dd_a, d_a, (size - 1) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_b, d_b, (size) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_c, d_c, (size - 1) * sizeof(double), hipMemcpyDeviceToDevice);
    hipMemcpy(dd_y, d_y, (size) * sizeof(double), hipMemcpyDeviceToDevice);

    // forward reduction
    for (int i = 0; i < depth; i++)
    {
        int thread_count = (size / pow(2, i + 1));
        int block_count = 1;
        if (thread_count > block_size) {
            block_count = ceil(thread_count / block_size);
            thread_count = block_size;
        }

        forward_reduce<<<block_count, thread_count>>>(dd_a, dd_b, dd_c, d_x, dd_y, i);
        hipDeviceSynchronize();
    }

    // elimination
    elimination<<<1, 1>>>(dd_a, dd_b, dd_c, d_x, dd_y);
    hipDeviceSynchronize();

    // backward subtitution
    for (int i = depth - 1; i >= 0; i--)
    {
        int thread_count = (size / pow(2, i + 1));
        int block_count = 1;
        if (thread_count > block_size) {
            block_count = ceil(thread_count / block_size);
            thread_count = block_size;
        }

        backward_subs << <block_count, thread_count >> > (dd_a, dd_b, dd_c, d_x, dd_y, i);
        hipDeviceSynchronize();
    }

    hipFree(dd_a);
    hipFree(dd_b);
    hipFree(dd_c);
    hipFree(dd_y);
}

__global__ void forward_reduce(double* a, double* b, double* c, double* x, double* y, int depth)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    int idx = ((1 << depth + 1) - 1) + (i * (1 << depth + 1));
    int offset = 1 << depth;

    if (idx == (1 << depth + 1) - 1)
    {
        double alpha = -a[idx - 1] / b[idx - offset];
        double beta = -c[idx] / b[idx + offset];
        a[idx - 1] = 0;
        b[idx] = b[idx] + alpha * c[idx - offset] + beta * a[idx + offset - 1];
        c[idx] = beta * c[idx + offset];
        y[idx] = y[idx] + alpha * y[idx - offset] + beta * y[idx + offset];
    }
    else if (idx == size - 1)
    {
        double alpha = -a[idx - 1] / b[idx - offset];
        a[idx - 1] = alpha * a[idx - offset - 1];
        b[idx] = b[idx] + alpha * c[idx - offset];
        y[idx] = y[idx] + alpha * y[idx - offset];
    }
    else
    {
        double alpha = -a[idx - 1] / b[idx - offset];
        double beta = -c[idx] / b[idx + offset];
        a[idx - 1] = alpha * a[idx - offset - 1];
        b[idx] = b[idx] + alpha * c[idx - offset] + beta * a[idx + offset - 1];
        c[idx] = beta * c[idx + offset];
        y[idx] = y[idx] + alpha * y[idx - offset] + beta * y[idx + offset];
    }
}

__global__ void elimination(double* a, double* b, double* c, double* x, double* y)
{
    // elimination to get X(n / 2 - 1) and X(n - 1)
    int idx_mid = int(size / 2) - 1;
    y[idx_mid] = y[idx_mid] / b[idx_mid];
    c[idx_mid] = c[idx_mid] / b[idx_mid];
    x[size - 1] = (y[size - 1] - (a[size - 2] * y[idx_mid])) / (b[size - 1] - (a[size - 2] * c[idx_mid]));
    x[idx_mid] = y[idx_mid] - c[idx_mid] * x[size - 1];
}

__global__ void backward_subs(double* a, double* b, double* c, double* x, double* y, int depth)
{
    int i = blockIdx.x * block_size + threadIdx.x;
    if (depth == 0)
    {
        int idx = i * 2;
        if (idx == 0)
        {
            x[idx] = (y[idx] - c[idx] * x[idx + 1]) / b[idx];
        }
        else
        {
            x[idx] = (y[idx] - a[idx - 1] * x[idx - 1] - c[idx] * x[idx + 1]) / b[idx];
        }
    }
    else
    {
        int idx = ((1 << depth) - 1) + i * (1 << depth + 1);
        int offset = (1 << depth);

        x[idx] = (y[idx] - a[idx - 1] * x[idx - offset] - c[idx] * x[idx + offset]) / b[idx];
    }
}
